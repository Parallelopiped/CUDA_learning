
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
const int size = 5;

__global__ void add(int *a, int *b, int *c){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = i + size * j;
    if(i < size && j < size){
        c[idx] = a[idx] + b[idx];
    }
}

int main(){
    int a[size][size];
    int b[size][size];
    int c[size][size];
    int *g_a, *g_b, *g_c;
    for(int i = 0; i < size; i++){
        for(int j = 0; j < size; j++){
            a[i][j] = 1;
            b[i][j] = 2;
        }
    }
    a[0][3] = 4;
    hipMalloc((void**)&g_a, sizeof(int) * size * size);
    hipMalloc((void**)&g_b, sizeof(int) * size * size);
    hipMalloc((void**)&g_c, sizeof(int) * size * size);
    hipMemcpy(g_a, a, sizeof(int) * size * size, hipMemcpyHostToDevice);
    hipMemcpy(g_b, b, sizeof(int) * size * size, hipMemcpyHostToDevice);
    //grid数量确保够用
    dim3 ThreadsPerBlock(16, 16);
    dim3 BlocksPerGrid((size - 1) / ThreadsPerBlock.x + 1, (size - 1) / ThreadsPerBlock.y + 1);
    add<<<BlocksPerGrid, ThreadsPerBlock>>>(g_a, g_b, g_c);
    //add<<<1, ThreadsPerBlock>>>(g_a, g_b, g_c);
    hipMemcpy(c, g_c, sizeof(int) * size * size, hipMemcpyDeviceToHost);
    for(int i = 0; i < size; i++){
        for(int j = 0; j < size; j++)
            printf("%d ", c[i][j]);
        puts("");
    }
    return 0;
}